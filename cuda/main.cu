#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "games/connect4/connect4.cuh"
#include "games/tictactoe/tictactoe.cuh"
#include "networks/gat/gat.cuh"
#include "self_play/self_play.cuh"
#include "utils/cuda_utils.cuh"
#include "config.h"  // Include the new configuration header

#define CUDA_CHECK(call) { \
    hipError_t status = call; \
    if (status != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(status)); \
        exit(1); \
    } \
}

int main(int argc, char* argv[]) {
    // Load configuration
    GlobalConfig* config = load_config("config.json");
    if (!config) {
        fprintf(stderr, "Failed to load configuration\n");
        return 1;
    }

    // Initialize CUDA
    CUDA_CHECK(hipSetDevice(0));

    // Parse command-line arguments
    const char* game_type = "tictactoe";
    if (argc > 1) {
        game_type = argv[1];
    }

    // Create game instance
    IGame* game = NULL;
    if (strcmp(game_type, "tictactoe") == 0) {
        game = (IGame*)create_tictactoe_game();
    } else if (strcmp(game_type, "connect4") == 0) {
        game = (IGame*)create_connect4_game();
    } else {
        fprintf(stderr, "Unsupported game type: %s\n", game_type);
        free_config(config);
        return 1;
    }

    if (!game) {
        fprintf(stderr, "Failed to create game instance\n");
        free_config(config);
        return 1;
    }

    // Create neural network instance
    INeuralNet* nnet = create_gat_model(game);
    if (!nnet) {
        fprintf(stderr, "Failed to create GAT neural network instance\n");
        game->destroy(game);
        free_config(config);
        return 1;
    }

    // Use the configuration for self-play
    SelfPlayConfig sp_config = config->self_play.config;

    // Create self-play pipeline
    SelfPlayPipeline* pipeline = create_self_play_pipeline(game, nnet, sp_config);
    if (!pipeline) {
        fprintf(stderr, "Failed to create self-play pipeline\n");
        nnet->destroy(nnet);
        game->destroy(game);
        free_config(config);
        return 1;
    }

    // Main training loop
    for (int i = 1; i <= sp_config.numIters; i++) {
        printf("Starting iteration %d\n", i);

        // Execute self-play
        execute_self_play(pipeline);

        // Train neural network
        learn(pipeline);

        // Optionally save checkpoint
        if (i % 10 == 0) {
            char filename[config->neural_network.max_filename_length];
            snprintf(filename, sizeof(filename), "checkpoint_%04d.pth", i);
            nnet->save_checkpoint(nnet, sp_config.checkpoint, filename);
            printf("Saved checkpoint: %s\n", filename);
        }
    }

    // Clean up
    destroy_self_play_pipeline(pipeline);
    nnet->destroy(nnet);
    game->destroy(game);
    free_config(config);

    // Reset CUDA device
    CUDA_CHECK(hipDeviceReset());

    printf("Training completed successfully\n");
    return 0;
}