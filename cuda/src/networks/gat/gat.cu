#include "gat.cuh"

#include <hiprand.h>
#include <torch/serialize.h>

static void gat_init(INeuralNet* self, const IGame* game) {
    GATWrapper* wrapper = (GATWrapper*)self;
    GATModel* model = &wrapper->model;

    // Initialize model configuration
    init_model_config(model, game);

    // Initialize cuDNN
    hipdnnCreate(&model->cudnn_handle);

    // Initialize input block
    init_input_block(model);

    // Initialize GAT layers
    init_gat_layers(model);

    // Initialize output block
    init_output_block(model);

    // Initialize weights
    init_weights(model);

    // Initialize PyTorch optimizer
    std::vector<torch::Tensor> params;
    // Add all weights and biases to params vector
    // Input block
    params.push_back(torch::from_blob(model->input_weights, {model->config.input_features, model->config.hidden_features}, torch::kCUDA));
    params.push_back(torch::from_blob(model->input_bias, {model->config.hidden_features}, torch::kCUDA));
    // GAT layers
    for (int i = 0; i < model->config.num_layers; i++) {
        params.push_back(torch::from_blob(model->layer_weights[i], {model->config.hidden_features, model->config.hidden_features}, torch::kCUDA));
        params.push_back(torch::from_blob(model->layer_biases[i], {model->config.hidden_features}, torch::kCUDA));
        params.push_back(torch::from_blob(model->attention_weights[i], {model->config.num_heads, 2, model->config.hidden_features}, torch::kCUDA));
    }
    // Output block
    params.push_back(torch::from_blob(model->value_weights, {model->config.hidden_features}, torch::kCUDA));
    params.push_back(torch::from_blob(model->value_bias, {1}, torch::kCUDA));
    params.push_back(torch::from_blob(model->policy_weights, {model->config.hidden_features, model->config.num_actions}, torch::kCUDA));
    params.push_back(torch::from_blob(model->policy_bias, {model->config.num_actions}, torch::kCUDA));

    model->optimizer = new torch::optim::Adam(params, torch::optim::AdamOptions(model->config.learning_rate).weight_decay(model->config.weight_decay));

    // Allocate workspace for cuDNN
    size_t workspace_size = 0;
    hipdnnGetConvolutionForwardWorkspaceSize(model->cudnn_handle, model->input_descriptor, model->layer_descriptors[0],
                                            /* convolution descriptor */, model->layer_descriptors[0],
                                            HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, &workspace_size);
    hipMalloc(&model->workspace, workspace_size);
    model->workspace_size = workspace_size;
}

INeuralNet* create_gat_model(const IGame* game) {
    GATWrapper* wrapper = (GATWrapper*)malloc(sizeof(GATWrapper));
    wrapper->base.impl = wrapper;
    wrapper->base.init = gat_init;
    wrapper->base.train = gat_train;
    wrapper->base.predict = gat_predict;
    wrapper->base.save_checkpoint = gat_save_checkpoint;
    wrapper->base.load_checkpoint = gat_load_checkpoint;
    wrapper->base.destroy = gat_destroy;

    gat_init(&wrapper->base, game);

    return &wrapper->base;
}

static void gat_save_checkpoint(INeuralNet* self, const char* folder, const char* filename) {
    GATWrapper* wrapper = (GATWrapper*)self;
    GATModel* model = &wrapper->model;

    char filepath[MAX_FILENAME_LENGTH];
    snprintf(filepath, MAX_FILENAME_LENGTH, "%s/%s", folder, filename);

    torch::serialize::OutputArchive archive;

    // Save model configuration
    archive.write("config", torch::from_blob(&model->config, {sizeof(ModelConfig)}, torch::kByte));

    // Save input block weights
    archive.write("input_weights", torch::from_blob(model->input_weights, 
        {model->config.input_features, model->config.hidden_features}, torch::kFloat32));
    archive.write("input_bias", torch::from_blob(model->input_bias, 
        {model->config.hidden_features}, torch::kFloat32));

    // Save GAT layer weights
    for (int i = 0; i < model->config.num_layers; i++) {
        char key[50];
        snprintf(key, sizeof(key), "layer_weights_%d", i);
        archive.write(key, torch::from_blob(model->layer_weights[i], 
            {model->config.hidden_features, model->config.hidden_features}, torch::kFloat32));
        
        snprintf(key, sizeof(key), "layer_biases_%d", i);
        archive.write(key, torch::from_blob(model->layer_biases[i], 
            {model->config.hidden_features}, torch::kFloat32));
        
        snprintf(key, sizeof(key), "attention_weights_%d", i);
        archive.write(key, torch::from_blob(model->attention_weights[i], 
            {model->config.num_heads, 2, model->config.hidden_features}, torch::kFloat32));
    }

    // Save output block weights
    archive.write("value_weights", torch::from_blob(model->value_weights, 
        {model->config.hidden_features}, torch::kFloat32));
    archive.write("value_bias", torch::from_blob(model->value_bias, {1}, torch::kFloat32));
    archive.write("policy_weights", torch::from_blob(model->policy_weights, 
        {model->config.hidden_features, model->config.num_actions}, torch::kFloat32));
    archive.write("policy_bias", torch::from_blob(model->policy_bias, 
        {model->config.num_actions}, torch::kFloat32));

    // Save optimizer state
    archive.write("optimizer", model->optimizer->state_dict());

    torch::serialize::save_to_file(archive, filepath);
}

static void gat_load_checkpoint(INeuralNet* self, const char* folder, const char* filename) {
    GATWrapper* wrapper = (GATWrapper*)self;
    GATModel* model = &wrapper->model;

    char filepath[MAX_FILENAME_LENGTH];
    snprintf(filepath, MAX_FILENAME_LENGTH, "%s/%s", folder, filename);

    torch::serialize::InputArchive archive;
    torch::serialize::load_from_file(archive, filepath);

    // Load model configuration
    torch::Tensor config_tensor;
    archive.read("config", config_tensor);
    memcpy(&model->config, config_tensor.data_ptr(), sizeof(ModelConfig));

    // Reallocate memory if necessary (in case the loaded model has different dimensions)
    // This part is omitted for brevity, but you should implement it in a production environment

    // Load input block weights
    torch::Tensor input_weights, input_bias;
    archive.read("input_weights", input_weights);
    archive.read("input_bias", input_bias);
    hipMemcpy(model->input_weights, input_weights.data_ptr(), 
        input_weights.numel() * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(model->input_bias, input_bias.data_ptr(), 
        input_bias.numel() * sizeof(float), hipMemcpyDeviceToDevice);

    // Load GAT layer weights
    for (int i = 0; i < model->config.num_layers; i++) {
        char key[50];
        torch::Tensor layer_weights, layer_biases, attention_weights;
        
        snprintf(key, sizeof(key), "layer_weights_%d", i);
        archive.read(key, layer_weights);
        hipMemcpy(model->layer_weights[i], layer_weights.data_ptr(), 
            layer_weights.numel() * sizeof(float), hipMemcpyDeviceToDevice);
        
        snprintf(key, sizeof(key), "layer_biases_%d", i);
        archive.read(key, layer_biases);
        hipMemcpy(model->layer_biases[i], layer_biases.data_ptr(), 
            layer_biases.numel() * sizeof(float), hipMemcpyDeviceToDevice);
        
        snprintf(key, sizeof(key), "attention_weights_%d", i);
        archive.read(key, attention_weights);
        hipMemcpy(model->attention_weights[i], attention_weights.data_ptr(), 
            attention_weights.numel() * sizeof(float), hipMemcpyDeviceToDevice);
    }

    // Load output block weights
    torch::Tensor value_weights, value_bias, policy_weights, policy_bias;
    archive.read("value_weights", value_weights);
    archive.read("value_bias", value_bias);
    archive.read("policy_weights", policy_weights);
    archive.read("policy_bias", policy_bias);
    hipMemcpy(model->value_weights, value_weights.data_ptr(), 
        value_weights.numel() * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(model->value_bias, value_bias.data_ptr(), 
        value_bias.numel() * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(model->policy_weights, policy_weights.data_ptr(), 
        policy_weights.numel() * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(model->policy_bias, policy_bias.data_ptr(), 
        policy_bias.numel() * sizeof(float), hipMemcpyDeviceToDevice);

    // Load optimizer state
    torch::serialize::OutputArchive optimizer_archive;
    archive.read("optimizer", optimizer_archive);
    model->optimizer->load_state_dict(optimizer_archive);

    // Recreate cuDNN descriptors
    hipdnnSetTensor4dDescriptor(model->input_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                               model->config.batch_size, 1, model->config.max_nodes, model->config.input_features);

    for (int i = 0; i < model->config.num_layers; i++) {
        hipdnnSetTensor4dDescriptor(model->layer_descriptors[i], HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                   model->config.batch_size, model->config.num_heads, model->config.max_nodes, model->config.hidden_features);
    }

    hipdnnSetTensor4dDescriptor(model->value_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                               model->config.batch_size, 1, 1, model->config.hidden_features);
    hipdnnSetTensor4dDescriptor(model->policy_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                               model->config.batch_size, 1, 1, model->config.num_actions);
}

static void gat_destroy(INeuralNet* self) {
    GATWrapper* wrapper = (GATWrapper*)self;
    GATModel* model = &wrapper->model;

    // Free input block resources
    hipdnnDestroyTensorDescriptor(model->input_descriptor);
    hipFree(model->input_weights);
    hipFree(model->input_bias);

    // Free GAT layer resources
    for (int i = 0; i < model->config.num_layers; i++) {
        hipdnnDestroyTensorDescriptor(model->layer_descriptors[i]);
        hipFree(model->layer_weights[i]);
        hipFree(model->layer_biases[i]);
        hipFree(model->attention_weights[i]);
    }
    free(model->layer_descriptors);
    free(model->layer_weights);
    free(model->layer_biases);
    free(model->attention_weights);

    // Free output block resources
    hipdnnDestroyTensorDescriptor(model->value_descriptor);
    hipdnnDestroyTensorDescriptor(model->policy_descriptor);
    hipFree(model->value_weights);
    hipFree(model->value_bias);
    hipFree(model->policy_weights);
    hipFree(model->policy_bias);

    // Free cuDNN workspace
    hipFree(model->workspace);

    // Destroy cuDNN handle
    hipdnnDestroy(model->cudnn_handle);

    // Delete PyTorch optimizer
    delete model->optimizer;

    // Free the wrapper itself
    free(wrapper);
}

/*************************************************************************************************************************************************************
 * INIT HELPER FUNCTIONS
**************************************************************************************************************************************************************/


static void init_model_config(GATModel* model, const IGame* game) {
    int board_size = game->get_board_size(game);
    model->config.input_features = board_size * board_size;  // Assuming square board
    model->config.hidden_features = 256;  // You can adjust this
    model->config.output_features = 256;  // You can adjust this
    model->config.num_heads = 8;  // Typical value, can be adjusted
    model->config.num_layers = 3;  // You can adjust this
    model->config.num_actions = game->get_action_size(game);
    model->config.max_nodes = board_size * board_size;
    model->config.max_edges = model->config.max_nodes * model->config.max_nodes;  // Fully connected graph
    model->config.learning_rate = 0.001f;
    model->config.weight_decay = 0.0001f;
    model->config.dropout = 0.1f;
    model->config.alpha = 0.2f;  // LeakyReLU angle
    model->config.batch_size = 64;  // You can adjust this
    model->config.epochs = 10;  // You can adjust this
}

static void init_input_block(GATModel* model) {
    hipdnnCreateTensorDescriptor(&model->input_descriptor);
    hipdnnSetTensor4dDescriptor(model->input_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                               model->config.batch_size, 1, model->config.max_nodes, model->config.input_features);

    hipMalloc(&model->input_weights, sizeof(float) * model->config.input_features * model->config.hidden_features);
    hipMalloc(&model->input_bias, sizeof(float) * model->config.hidden_features);
}

static void init_gat_layers(GATModel* model) {
    model->layer_descriptors = (hipdnnTensorDescriptor_t*)malloc(model->config.num_layers * sizeof(hipdnnTensorDescriptor_t));
    model->layer_weights = (float**)malloc(model->config.num_layers * sizeof(float*));
    model->layer_biases = (float**)malloc(model->config.num_layers * sizeof(float*));
    model->attention_weights = (float**)malloc(model->config.num_layers * sizeof(float*));

    for (int i = 0; i < model->config.num_layers; i++) {
        hipdnnCreateTensorDescriptor(&model->layer_descriptors[i]);
        hipdnnSetTensor4dDescriptor(model->layer_descriptors[i], HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                   model->config.batch_size, model->config.num_heads, model->config.max_nodes, model->config.hidden_features);

        hipMalloc(&model->layer_weights[i], sizeof(float) * model->config.hidden_features * model->config.hidden_features);
        hipMalloc(&model->layer_biases[i], sizeof(float) * model->config.hidden_features);
        hipMalloc(&model->attention_weights[i], sizeof(float) * model->config.num_heads * 2 * model->config.hidden_features);
    }
}

static void init_output_block(GATModel* model) {
    hipdnnCreateTensorDescriptor(&model->value_descriptor);
    hipdnnCreateTensorDescriptor(&model->policy_descriptor);

    hipdnnSetTensor4dDescriptor(model->value_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                               model->config.batch_size, 1, 1, model->config.hidden_features);
    hipdnnSetTensor4dDescriptor(model->policy_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                               model->config.batch_size, 1, 1, model->config.num_actions);

    hipMalloc(&model->value_weights, sizeof(float) * model->config.hidden_features);
    hipMalloc(&model->value_bias, sizeof(float));
    hipMalloc(&model->policy_weights, sizeof(float) * model->config.hidden_features * model->config.num_actions);
    hipMalloc(&model->policy_bias, sizeof(float) * model->config.num_actions);
}

static void init_weights(GATModel* model) {
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));

    // Initialize input weights
    hiprandGenerateNormal(gen, model->input_weights, model->config.input_features * model->config.hidden_features, 0, 0.1);
    hiprandGenerateNormal(gen, model->input_bias, model->config.hidden_features, 0, 0.1);

    // Initialize GAT layer weights
    for (int i = 0; i < model->config.num_layers; i++) {
        hiprandGenerateNormal(gen, model->layer_weights[i], model->config.hidden_features * model->config.hidden_features, 0, 0.1);
        hiprandGenerateNormal(gen, model->layer_biases[i], model->config.hidden_features, 0, 0.1);
        hiprandGenerateNormal(gen, model->attention_weights[i], model->config.num_heads * 2 * model->config.hidden_features, 0, 0.1);
    }

    // Initialize output weights
    hiprandGenerateNormal(gen, model->value_weights, model->config.hidden_features, 0, 0.1);
    hiprandGenerateNormal(gen, model->value_bias, 1, 0, 0.1);
    hiprandGenerateNormal(gen, model->policy_weights, model->config.hidden_features * model->config.num_actions, 0, 0.1);
    hiprandGenerateNormal(gen, model->policy_bias, model->config.num_actions, 0, 0.1);

    hiprandDestroyGenerator(gen);
}