#include "hip/hip_runtime.h"
#include "self_play.cuh"

#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

SelfPlayPipeline* create_self_play_pipeline(IGame* game, INeuralNet* nnet, SelfPlayConfig config) {
    SelfPlayPipeline* pipeline = (SelfPlayPipeline*)malloc(sizeof(SelfPlayPipeline));
    if (!pipeline) {
        fprintf(stderr, "Failed to allocate memory for SelfPlayPipeline\n");
        return NULL;
    }

    pipeline->game = game;
    pipeline->nnet = nnet;
    pipeline->config = config;

    // Initialize MCTS
    pipeline->mcts = mcts_init(game);
    if (!pipeline->mcts) {
        fprintf(stderr, "Failed to initialize MCTS\n");
        free(pipeline);
        return NULL;
    }

    // Allocate GPU resources
    CUDA_CHECK(hipMalloc(&pipeline->d_rng_states, config.numGames * sizeof(hiprandState)));
    CUDA_CHECK(hipMalloc(&pipeline->d_boards, config.numGames * MAX_BOARD_SIZE * sizeof(int)));
    CUDA_CHECK(hipMalloc(&pipeline->d_pis, config.numGames * MAX_BOARD_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&pipeline->d_vs, config.numGames * sizeof(float)));
    CUDA_CHECK(hipMalloc(&pipeline->d_mcts_nodes, config.numGames * sizeof(MCTSNode)));

    // Initialize d_examples to nullptr
    pipeline->d_examples = nullptr;

    // Initialize RNG states
    init_rng<<<(config.numGames + 255) / 256, 256>>>(pipeline->d_rng_states, time(NULL));
    CUDA_CHECK(hipGetLastError());

    // Allocate CPU resources for training examples history
    pipeline->trainExamplesHistory = (TrainingExample**)malloc(config.numItersForTrainExamplesHistory * sizeof(TrainingExample*));
    if (!pipeline->trainExamplesHistory) {
        fprintf(stderr, "Failed to allocate memory for trainExamplesHistory\n");
        destroy_self_play_pipeline(pipeline);
        return NULL;
    }
    pipeline->historySize = 0;
    pipeline->skipFirstSelfPlay = false;

    return pipeline;
}

void destroy_self_play_pipeline(SelfPlayPipeline* pipeline) {
    if (!pipeline) return;

    mcts_free(pipeline->mcts);

    CUDA_CHECK(hipFree(pipeline->d_rng_states));
    CUDA_CHECK(hipFree(pipeline->d_boards));
    CUDA_CHECK(hipFree(pipeline->d_pis));
    CUDA_CHECK(hipFree(pipeline->d_vs));
    CUDA_CHECK(hipFree(pipeline->d_mcts_nodes));

    // Free memory allocated for d_examples
    if (pipeline->d_examples) {
        CUDA_CHECK(hipFree(pipeline->d_examples));
    }

    for (int i = 0; i < pipeline->historySize; i++) {
        free(pipeline->trainExamplesHistory[i]);
    }
    free(pipeline->trainExamplesHistory);

    free(pipeline);
}


void execute_self_play(SelfPlayPipeline* pipeline) {
    int numGames = pipeline->config.numGames;
    int numMCTSSims = pipeline->config.numMCTSSims;
    int tempThreshold = pipeline->config.tempThreshold;
    
    // Initialize boards on GPU
    thrust::host_vector<int> h_init_board(MAX_BOARD_SIZE);
    pipeline->game->get_init_board(pipeline->game, h_init_board.data());
    thrust::device_vector<int> d_boards(numGames * MAX_BOARD_SIZE, 0);
    for (int i = 0; i < numGames; ++i) {
        thrust::copy(h_init_board.begin(), h_init_board.end(), d_boards.begin() + i * MAX_BOARD_SIZE);
    }

    // Initialize MCTS nodes
    thrust::device_vector<MCTSNode> d_mcts_roots(numGames);

    // Initialize other necessary arrays
    thrust::device_vector<float> d_pis(numGames * MAX_BOARD_SIZE);
    thrust::device_vector<float> d_vs(numGames);
    thrust::device_vector<int> d_players(numGames, 1);  // Start with player 1 for all games

    // Allocate memory for examples if not already allocated
    if (pipeline->d_examples == nullptr) {
        CUDA_CHECK(hipMalloc(&pipeline->d_examples, numGames * MAX_GAME_LENGTH * sizeof(TrainingExample)));
    }

    // Launch parallel self-play kernel
    dim3 grid((numGames + 255) / 256, 1, 1);
    dim3 block(256, 1, 1);
    
    parallel_self_play_kernel<<<grid, block>>>(
        thrust::raw_pointer_cast(d_mcts_roots.data()),
        thrust::raw_pointer_cast(d_boards.data()),
        thrust::raw_pointer_cast(d_pis.data()),
        thrust::raw_pointer_cast(d_vs.data()),
        thrust::raw_pointer_cast(d_players.data()),
        pipeline->d_rng_states,
        pipeline->game,
        pipeline->nnet,
        numGames,
        numMCTSSims,
        tempThreshold,
        pipeline->d_examples
    );
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy results back to CPU and process
    thrust::host_vector<TrainingExample> h_examples(numGames * MAX_GAME_LENGTH);
    CUDA_CHECK(hipMemcpy(h_examples.data(), pipeline->d_examples, numGames * MAX_GAME_LENGTH * sizeof(TrainingExample), hipMemcpyDeviceToHost));

    // Process and store examples
    int totalExamples = 0;
    for (int i = 0; i < numGames; i++) {
        for (int j = 0; j < MAX_GAME_LENGTH; j++) {
            if (h_examples[i * MAX_GAME_LENGTH + j].board[0] == TERMINAL_STATE) {
                break;
            }
            totalExamples++;
        }
    }

    // Add examples to the training history
    add_to_training_history(pipeline, h_examples.data(), totalExamples);
}

__global__ void init_rng(hiprandState* states, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

__global__ void parallel_self_play_kernel(
    MCTSNode* roots, int* boards, float* pis, float* vs, int* players,
    hiprandState* rng_states, IGame* game, INeuralNet* nnet,
    int num_games, int num_mcts_sims, int temp_threshold,
    TrainingExample* examples
) {
    int game_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (game_idx >= num_games) return;

    hiprandState* rng_state = &rng_states[game_idx];
    MCTSNode* root = &roots[game_idx];
    int* board = &boards[game_idx * MAX_BOARD_SIZE];
    float* pi = &pis[game_idx * MAX_BOARD_SIZE];
    int player = players[game_idx];
    int moves = 0;

    while (true) {
        // Perform MCTS simulations
        for (int i = 0; i < num_mcts_sims; i++) {
            mcts_simulate(root, board, player, rng_state, game, nnet);
        }

        // Compute policy from visit counts
        float temp = (moves < temp_threshold) ? 1.0f : 1e-3f;
        mcts_get_policy(root, pi, temp);

        // Store the current state as a training example
        TrainingExample* example = &examples[game_idx * MAX_GAME_LENGTH + moves];
        memcpy(example->board, board, MAX_BOARD_SIZE * sizeof(int));
        memcpy(example->pi, pi, MAX_BOARD_SIZE * sizeof(float));

        // Select action
        int action = select_action(pi, game->get_action_size(game), rng_state);

        // Apply action
        int next_board[MAX_BOARD_SIZE];
        int next_player;
        game->get_next_state_cuda(game, board, player, action, next_board, &next_player);

        // Check if game has ended
        float reward = game->get_game_ended_cuda(game, next_board, next_player);
        if (reward != 0) {
            // Game has ended, update all examples with the reward
            for (int i = 0; i <= moves; i++) {
                TrainingExample* ex = &examples[game_idx * MAX_GAME_LENGTH + i];
                ex->v = reward * ((i % 2 == 0) ? 1 : -1);
            }
            vs[game_idx] = reward;
            break;
        }

        // Move to next state
        memcpy(board, next_board, MAX_BOARD_SIZE * sizeof(int));
        player = next_player;
        root = mcts_move_to_child(root, action);
        moves++;

        if (moves >= MAX_GAME_LENGTH - 1) {
            // Force end of game if it's taking too long
            for (int i = 0; i <= moves; i++) {
                TrainingExample* ex = &examples[game_idx * MAX_GAME_LENGTH + i];
                ex->v = 0.0f;  // Draw
            }
            vs[game_idx] = 0.0f;
            break;
        }
    }

    players[game_idx] = player;  // Update final player state
}

// Helper functions

void add_to_training_history(SelfPlayPipeline* pipeline, TrainingExample* examples, int num_examples) {
    // If the history is full, remove the oldest entry
    if (pipeline->historySize >= pipeline->config.numItersForTrainExamplesHistory) {
        free(pipeline->trainExamplesHistory[0]);
        memmove(pipeline->trainExamplesHistory, pipeline->trainExamplesHistory + 1,
                (pipeline->historySize - 1) * sizeof(TrainingExample*));
        pipeline->historySize--;
    }

    // Allocate memory for the new examples
    TrainingExample* new_examples = (TrainingExample*)malloc(num_examples * sizeof(TrainingExample));
    if (new_examples == nullptr) {
        fprintf(stderr, "Failed to allocate memory for new training examples\n");
        return;
    }

    // Copy the examples
    memcpy(new_examples, examples, num_examples * sizeof(TrainingExample));

    // Add the new examples to the history
    pipeline->trainExamplesHistory[pipeline->historySize] = new_examples;
    pipeline->historySize++;
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

void save_train_examples(SelfPlayPipeline* pipeline, int iteration) {
    char filename[256];
    snprintf(filename, sizeof(filename), "%s/train_examples_%d.bin", pipeline->config.checkpoint, iteration);

    FILE* file = fopen(filename, "wb");
    if (file == NULL) {
        fprintf(stderr, "Error: Unable to open file %s for writing\n", filename);
        return;
    }

    // Write the number of iterations in the history
    fwrite(&pipeline->historySize, sizeof(int), 1, file);

    // Write each iteration's examples
    for (int i = 0; i < pipeline->historySize; i++) {
        // Write the number of examples in this iteration
        int num_examples = pipeline->config.numEps * MAX_GAME_LENGTH;
        fwrite(&num_examples, sizeof(int), 1, file);

        // Write the examples
        fwrite(pipeline->trainExamplesHistory[i], sizeof(TrainingExample), num_examples, file);
    }

    fclose(file);
    printf("Training examples saved to %s\n", filename);
}

void load_train_examples(SelfPlayPipeline* pipeline) {
    char filename[256];
    snprintf(filename, sizeof(filename), "%s/train_examples_latest.bin", pipeline->config.checkpoint);

    FILE* file = fopen(filename, "rb");
    if (file == NULL) {
        fprintf(stderr, "Error: Unable to open file %s for reading\n", filename);
        return;
    }

    // Read the number of iterations in the history
    int loaded_history_size;
    fread(&loaded_history_size, sizeof(int), 1, file);

    // Clear existing history
    for (int i = 0; i < pipeline->historySize; i++) {
        free(pipeline->trainExamplesHistory[i]);
    }
    pipeline->historySize = 0;

    // Read each iteration's examples
    for (int i = 0; i < loaded_history_size; i++) {
        // Read the number of examples in this iteration
        int num_examples;
        fread(&num_examples, sizeof(int), 1, file);

        // Allocate memory for the examples
        TrainingExample* examples = (TrainingExample*)malloc(num_examples * sizeof(TrainingExample));
        if (examples == NULL) {
            fprintf(stderr, "Error: Unable to allocate memory for loading examples\n");
            fclose(file);
            return;
        }

        // Read the examples
        fread(examples, sizeof(TrainingExample), num_examples, file);

        // Add to history
        pipeline->trainExamplesHistory[pipeline->historySize] = examples;
        pipeline->historySize++;
    }

    fclose(file);
    printf("Training examples loaded from %s\n", filename);
}

// MCTS and action selection helpers

__device__ void mcts_simulate(MCTSNode* node, int* board, int player, hiprandState* rng_state, IGame* game, INeuralNet* nnet) {
    if (game->get_game_ended_cuda(game, board, player) != 0) {
        // Game has ended, backpropagate the result
        mcts_backpropagate(node, -game->get_game_ended_cuda(game, board, player));
        return;
    }

    if (node->num_children == 0) {
        // Expand the node
        mcts_expand(node, board, player, game);

        // Evaluate the position using the neural network
        float value;
        float policy[MAX_BOARD_SIZE];
        nnet->predict(nnet, board, policy, &value);

        // Update node with the evaluation results
        for (int i = 0; i < node->num_children; i++) {
            node->P[i] = policy[node->children[i]->action];
        }

        // Backpropagate the value
        mcts_backpropagate(node, value);
    } else {
        // Select the best child according to the UCT formula
        MCTSNode* best_child = mcts_select_uct(node);

        // Recursively simulate from the best child
        int next_board[MAX_BOARD_SIZE];
        int next_player;
        game->get_next_state_cuda(game, board, player, best_child->action, next_board, &next_player);
        mcts_simulate(best_child, next_board, next_player, rng_state, game, nnet);
    }
}

__device__ void mcts_get_policy(MCTSNode* node, float* policy, float temperature) {
    int action_size = node->num_children;
    float sum = 0.0f;

    for (int i = 0; i < action_size; i++) {
        if (temperature == 0.0f) {
            policy[i] = (i == argmax(node->N, action_size)) ? 1.0f : 0.0f;
        } else {
            policy[i] = __powf(node->N[i], 1.0f / temperature);
        }
        sum += policy[i];
    }

    // Normalize the policy
    for (int i = 0; i < action_size; i++) {
        policy[i] /= sum;
    }
}

// Helper function to find the index of the maximum value
__device__ int argmax(float* arr, int size) {
    int max_idx = 0;
    for (int i = 1; i < size; i++) {
        if (arr[i] > arr[max_idx]) {
            max_idx = i;
        }
    }
    return max_idx;
}

__device__ int select_action(float* policy, int action_size, hiprandState* rng_state) {
    float r = hiprand_uniform(rng_state);
    float sum = 0.0f;
    for (int i = 0; i < action_size; i++) {
        sum += policy[i];
        if (r <= sum) {
            return i;
        }
    }
    // In case of rounding errors, return the last action
    return action_size - 1;
}

__device__ MCTSNode* mcts_move_to_child(MCTSNode* node, int action) {
    for (int i = 0; i < node->num_children; i++) {
        if (node->children[i]->action == action) {
            return node->children[i];
        }
    }
    // This should never happen if the action is valid
    return nullptr;
}